#include "hip/hip_runtime.h"
/*********************************************************************************/
/* Matrix product program for a multi-core CPU and for a many-core GPU           */
/* S. Vialle - November 2022                                                     */
/*********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "main.h"
#include "gpu.h"


/*-------------------------------------------------------------------------------*/
/* GPU symbols and global vars                                                   */
/*-------------------------------------------------------------------------------*/

// Symbols used by all kernels (Matrices on GPU devices) ------------------------
__device__ T_real GPU_A[SIZE][SIZE];
__device__ T_real GPU_B[SIZE][SIZE];
__device__ T_real GPU_C[SIZE][SIZE];

// Symbol and vars to call Cublas lib. ------------------------------------------
__device__ T_real GPU_Ctmp[SIZE][SIZE];   // New matrix buffer

T_real *AdrGPU_A = NULL;                  // Adresses of the symbols
T_real *AdrGPU_B = NULL;
T_real *AdrGPU_C = NULL;
T_real *AdrGPU_Ctmp = NULL; 

hipblasHandle_t cublasHandle;              // Handle on the Cublas lib.


/*-------------------------------------------------------------------------------*/
/* Init and finalize the GPU device.                                             */
/*-------------------------------------------------------------------------------*/
void gpuInit(void)
{
  // Init of the GPU device -----------------------------------------------------
  hipInit(0);
  
  // Turn CPU arrays A, B and C into "locked" memory areas to speedup transfers--
  CHECK_CUDA_SUCCESS(hipHostRegister(A,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(B,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(C,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the C CPU array");
  
  // Initializations to call Cublas lib. ----------------------------------------
  
  // - Extract address of GPU matrix "symbols" - useful when calling cuBLAS
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_A,GPU_A),"GPU_A adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_B,GPU_B),"GPU_B adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_C,GPU_C),"GPU_C adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_Ctmp,GPU_Ctmp),"GPU_Ctmp adr extraction");
  
  // - Initialize CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasCreate(&cublasHandle), "Init of the CUBLAS lib handle"); 
}


void gpuFinalize(void)
{
  // Turn "pinned (or locked)" CPU arrays into std array ------------------------
  CHECK_CUDA_SUCCESS(hipHostUnregister(A),
                     "Turning into std memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(B),
                     "Turning into std memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(C),
                     "Turning into std memory the C CPU array");

  // Free CUBLAS lib usage ------------------------------------------------------
  CHECK_CUBLAS_SUCCESS(hipblasDestroy(cublasHandle), "Free the CUBLAS lib");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                   */
/*-------------------------------------------------------------------------------*/
void gpuSetDataOnGPU(void)
{
  // Set GPU_A symbol
  CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(GPU_A), &A, sizeof(A), 0, hipMemcpyHostToDevice),
                    "Transfer A-->GPU_A");

  // Set GPU_B symbol
  CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(GPU_B), &B, sizeof(B), 0, hipMemcpyHostToDevice),
                    "Transfer B-->GPU_B");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                        */
/*-------------------------------------------------------------------------------*/
void gpuGetResultOnCPU(void)
{
  // Get GPU_C symbol
  CHECK_CUDA_SUCCESS(hipMemcpyFromSymbol(&C, HIP_SYMBOL(GPU_C), sizeof(C), 0, hipMemcpyDeviceToHost),
                    "Transfer C<--GPU_C");
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 1D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v0(void)
{
  // Index computations
  int row = blockIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_X_K0 + threadIdx.x;
  T_real res = 0.0;

  // Matrix product computation
  for (int k = 0; k < SIZE; k++) {
    res += GPU_A[row][k] * GPU_B[k][col];
  }
  GPU_C[row][col] = res;
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 2D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v1(void)
{
  // Index computations
  int row = blockIdx.y*BLOCK_SIZE_Y_K1 + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_X_K1 + threadIdx.x;
  T_real res = 0.0;

  // Matrix product computation
  for (int k = 0; k < SIZE; k++) {
    res += GPU_A[row][k] * GPU_B[k][col];
  }
  GPU_C[row][col] = res;
}

/*-------------------------------------------------------------------------------*/
/* Kernel K2                                                                     */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v2(void)
{
  int NbStep = gridDim.x;

  int row = blockIdx.y*BLOCK_SIZE_XY_K2 + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_XY_K2 + threadIdx.x;

  //Declaration of variables in Shared Memory
  __shared__ T_real shA[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  __shared__ T_real shB[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  __shared__ T_real shC[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  memset(shC, 0, sizeof(T_real)*BLOCK_SIZE_XY_K2*BLOCK_SIZE_XY_K2);

  //Computation loop (with synchronizations)
  for (int k = 0; k < NbStep; k++) {
    //Caching of data
    shA[threadIdx.y][threadIdx.x] = GPU_A[row][k*BLOCK_SIZE_XY_K2 + threadIdx.x];
    shB[threadIdx.y][threadIdx.x] = GPU_B[k*BLOCK_SIZE_XY_K2 + threadIdx.y][col];

    //Synchronization
    __syncthreads();

    //Computation of shared C
    for (int l=0; l< BLOCK_SIZE_XY_K2; l++) {
      shC[threadIdx.y][threadIdx.x] = shA[threadIdx.y][l] * shB[l][threadIdx.x];
    }

    __syncthreads();
  }

  //Storing results in global memory
  GPU_C[row][col] = shC[threadIdx.y][threadIdx.x];
}
  // successive threads in a warp access successive memory locations, it is coalesced

  // each thread makes NbStep*(1 access to GPU_A + 1 access to GPU_B) + 1 access to C = 2NbStep + 1
  // so for all threads (N² threads), it gives N² * (2NbStep + 1) = N² * (2N/BSX) + 1 

  // TD5 = 2N^3/BSX, et TD4: (2N + 1).N² et TD5 = TD4/BSXY
  // NbStep lecture rateau de A/NbStep lecture rateau de B 

  // TD5 = (2N + 1)/BSXY

  // Ws = Warp Size
  // en TD4, on avait N²/Ws * (2N + 1)
  // en TD5, on a réussi à diviser par BSXY

/*-------------------------------------------------------------------------------*/
/* Kernel K3                                                                     */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v3(void)
{
  //Declaration of variables in Shared Memory
  __shared__ float shA[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  __shared__ float shB[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  __shared__ float shC[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];

  // Index computations
  int row = blockIdx.y*BLOCK_SIZE_XY_K2 + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_XY_K2 + threadIdx.x;
  int NbStep = SIZE/BLOCK_SIZE_XY_K2;

  //Computation loop (with synchronizations)
  for (int k = 0; k < NbStep; k++) {
    //Caching of data
    if (row < SIZE && k*BLOCK_SIZE_XY_K2 + threadIdx.x < SIZE)
      shA[threadIdx.y][threadIdx.x] = GPU_A[row][k*BLOCK_SIZE_XY_K2 + threadIdx.x]; // !
    else
      shA[threadIdx.y][threadIdx.x] = 0;
    if (col < SIZE && k*BLOCK_SIZE_XY_K2 + threadIdx.y < SIZE)
      shB[threadIdx.y][threadIdx.x] = GPU_B[k*BLOCK_SIZE_XY_K2 + threadIdx.y][col]; // !
    else
      shB[threadIdx.y][threadIdx.x] = 0;
    //Synchronization
    __syncthreads();
    if (row < SIZE && col < SIZE) {
        //Computation of shared C
        for (int l=0; l< BLOCK_SIZE_XY_K2; l++) { // !
          shC[threadIdx.y][threadIdx.x] = shA[threadIdx.y][l] * shB[l][threadIdx.x];
        }
    }



    __syncthreads();
  }

  if (row < SIZE && col < SIZE) {
    //Storing results in global memory
    GPU_C[row][col] = shC[threadIdx.y][threadIdx.x]; // !
  }
}

/*-------------------------------------------------------------------------------*/
/* Transposition kernel using global memory and registers (slow version)         */
/*-------------------------------------------------------------------------------*/
__global__ void TransposeKernel_v0(T_real *MT, T_real *M, int mRow, int nCol)
{
 int row = threadIdx.y + blockIdx.y*BLOCK_SIZE_XY_KT0;
 int col = threadIdx.x + blockIdx.x*BLOCK_SIZE_XY_KT0;
 
 if (row < mRow && col < nCol)
   MT[col*mRow + row] = M[row*nCol + col];
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU.                                        */
/*-------------------------------------------------------------------------------*/
void gpuProduct(gkid_t kid)
{
 dim3 Dg = {0,0,0};   // Grid descriptor
 dim3 Db = {0,0,0};   // Block descriptor
 
 //T_real alpha;      // When using CUBLAS
 //T_real beta;       // When using CUBLAS

 switch(kid) {

 case GK0 : // Kernel v0 - 1D kernel using only resgisters and cache with generic matrix size
   // - init the grid of blocs
   Db.x = BLOCK_SIZE_X_K0;
   Db.y = 1;
   Db.z = 1;
   Dg.x = SIZE/BLOCK_SIZE_X_K0;
   Dg.y = SIZE;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v0<<<Dg,Db>>>();
   break;

 case GK1 : // kernel v1 : 2D kernel using only registers and cache with generic matrix size
   // - init the grid of blocs
   Db.x = BLOCK_SIZE_X_K1;
   Db.y = BLOCK_SIZE_Y_K1;
   Db.z = 1;
   Dg.x = SIZE/BLOCK_SIZE_X_K1;
   Dg.y = SIZE/BLOCK_SIZE_Y_K1;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v1<<<Dg,Db>>>();
   break;

 case GK2 : // kernel v2 : 2D kernel using the shared memories
   // - init the grid of blocs
   Db.x = BLOCK_SIZE_XY_K2;
   Db.y = BLOCK_SIZE_XY_K2;
   Db.z = 1;
   Dg.x = SIZE/BLOCK_SIZE_XY_K2;
   Dg.y = SIZE/BLOCK_SIZE_XY_K2;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v2<<<Dg,Db>>>();
   break;
  
 case GK3 : // kernel v3 : 2D kernel using the shared memories with generic matrix size
   break;

 case GK4 : // calling cublas gemm & user-defined transpose kernel
   break;
   
 case GK5 : // Calling cublas gemm & cublas geam kernels
   break;

 case GK6 : // Calling cublas gemm, using matrix math properties
   break;

 case GK7 : // Calling cublas gemmEx with std 32F datatypes, using Tensor cores
   break;

 case GK8 : // Calling cublas gemmEx with low precision on 32 bits datatypes, using Tensor cores
   break;

 default :
   fprintf(stderr,"Unknown GPU kernel!");
   exit(EXIT_FAILURE);
 } // End of switch
}




