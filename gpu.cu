#include "hip/hip_runtime.h"
/*********************************************************************************/
/* Matrix product program for a multi-core CPU and for a many-core GPU           */
/* S. Vialle - November 2022                                                     */
/*********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "main.h"
#include "gpu.h"


/*-------------------------------------------------------------------------------*/
/* GPU symbols and global vars                                                   */
/*-------------------------------------------------------------------------------*/

// Symbols used by all kernels (Matrices on GPU devices) ------------------------
__device__ T_real GPU_A[SIZE][SIZE];
__device__ T_real GPU_B[SIZE][SIZE];
__device__ T_real GPU_C[SIZE][SIZE];

// Symbol and vars to call Cublas lib. ------------------------------------------
__device__ T_real GPU_Ctmp[SIZE][SIZE];   // New matrix buffer

T_real *AdrGPU_A = NULL;                  // Adresses of the symbols
T_real *AdrGPU_B = NULL;
T_real *AdrGPU_C = NULL;
T_real *AdrGPU_Ctmp = NULL; 

hipblasHandle_t cublasHandle;              // Handle on the Cublas lib.


/*-------------------------------------------------------------------------------*/
/* Init and finalize the GPU device.                                             */
/*-------------------------------------------------------------------------------*/
void gpuInit(void)
{
  // Init of the GPU device -----------------------------------------------------
  hipInit(0);
  
  // Turn CPU arrays A, B and C into "locked" memory areas to speedup transfers--
  CHECK_CUDA_SUCCESS(hipHostRegister(A,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(B,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostRegister(C,SIZE*SIZE*sizeof(T_real),
                                      hipHostRegisterPortable),
                     "Turning into pinned memory the C CPU array");
  
  // Initializations to call Cublas lib. ----------------------------------------
  
  // - Extract address of GPU matrix "symbols" - useful when calling cuBLAS
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_A,GPU_A),"GPU_A adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_B,GPU_B),"GPU_B adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_C,GPU_C),"GPU_C adr extraction");
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **)&AdrGPU_Ctmp,GPU_Ctmp),"GPU_Ctmp adr extraction");
  
  // - Initialize CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasCreate(&cublasHandle), "Init of the CUBLAS lib handle"); 
}


void gpuFinalize(void)
{
  // Turn "pinned (or locked)" CPU arrays into std array ------------------------
  CHECK_CUDA_SUCCESS(hipHostUnregister(A),
                     "Turning into std memory the A CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(B),
                     "Turning into std memory the B CPU array");
  CHECK_CUDA_SUCCESS(hipHostUnregister(C),
                     "Turning into std memory the C CPU array");

  // Free CUBLAS lib usage ------------------------------------------------------
  CHECK_CUBLAS_SUCCESS(hipblasDestroy(cublasHandle), "Free the CUBLAS lib");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                   */
/*-------------------------------------------------------------------------------*/
void gpuSetDataOnGPU(void)
{
  // Set GPU_A symbol
  CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(GPU_A), &A, sizeof(A), 0, hipMemcpyHostToDevice),
                    "Transfer A-->GPU_A");

  // Set GPU_B symbol
  CHECK_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(GPU_B), &B, sizeof(B), 0, hipMemcpyHostToDevice),
                    "Transfer B-->GPU_B");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                        */
/*-------------------------------------------------------------------------------*/
void gpuGetResultOnCPU(void)
{
  // Get GPU_C symbol
  CHECK_CUDA_SUCCESS(hipMemcpyFromSymbol(&C, HIP_SYMBOL(GPU_C), sizeof(C), 0, hipMemcpyDeviceToHost),
                    "Transfer C<--GPU_C");
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 1D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v0(void)
{
  // Index computations
  int row = blockIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_X_K0 + threadIdx.x;
  T_real res = 0.0;

  if (col < SIZE) {
    // Matrix product computation
    for (int k = 0; k < SIZE; k++) {
      res += GPU_A[row][k] * GPU_B[k][col];
    }
    GPU_C[row][col] = res;
  }
  
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU - 2D & generic matrix size              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v1(void)
{
  // Index computations
  int row = blockIdx.y*BLOCK_SIZE_Y_K1 + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_X_K1 + threadIdx.x;
  T_real res = 0.0;

  if ((row < SIZE) && (col < SIZE)){
    // Matrix product computation
    for (int k = 0; k < SIZE; k++) {
      res += GPU_A[row][k] * GPU_B[k][col];
    }
    GPU_C[row][col] = res;
  }
  
}


/*-------------------------------------------------------------------------------*/
/* Shared memory caching - 2D & matrix size is multiple of block                 */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v2(void)
{
  // Index computations
  int nbSteps = gridDim.x;

  int row = blockIdx.y*BLOCK_SIZE_XY_K2 + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_XY_K2 + threadIdx.x;

  // Shared memory arrays
  __shared__ T_real shared_A_block[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  __shared__ T_real shared_B_block[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  __shared__ T_real shared_C_block[BLOCK_SIZE_XY_K2][BLOCK_SIZE_XY_K2];
  shared_C_block[threadIdx.y][threadIdx.x] = 0;
<<<<<<< HEAD

=======
>>>>>>> f6677404b98f5ea4f868b862a629c08771875253
  
  // Matrix product computation
  for (int step = 0; step < nbSteps; step++) {
    // RAM to shared memory
    shared_A_block[threadIdx.y][threadIdx.x] = GPU_A[row][step * BLOCK_SIZE_XY_K2 + threadIdx.x];
    shared_B_block[threadIdx.y][threadIdx.x] = GPU_B[step * BLOCK_SIZE_XY_K2 + threadIdx.y][col];

    __syncthreads();
    // Partial matrix product
    for (int k = 0; k < BLOCK_SIZE_XY_K2; k++) {
      shared_C_block[threadIdx.y][threadIdx.x] += shared_A_block[threadIdx.y][k] * shared_B_block[k][threadIdx.x];
    }
    __syncthreads();
  }

  GPU_C[row][col] = shared_C_block[threadIdx.y][threadIdx.x];
}


/*-------------------------------------------------------------------------------*/
/* Shared memory caching - 2D & generic matrix size                              */
/*-------------------------------------------------------------------------------*/
__global__ void MatrixProductKernel_v3(void)
{
  int nbSteps = gridDim.x;
  int row = blockIdx.y*BLOCK_SIZE_XY_K3 + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_XY_K3 + threadIdx.x;

  // Shared memory arrays
  __shared__ T_real shared_A_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  __shared__ T_real shared_B_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  __shared__ T_real shared_C_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  shared_C_block[threadIdx.y][threadIdx.x] = 0;

  // Matrix product computation
  for (int step = 0; step < nbSteps; step++) {
    // RAM to shared memory
    if (row < SIZE && (step * BLOCK_SIZE_XY_K3 + threadIdx.x < SIZE))
      shared_A_block[threadIdx.y][threadIdx.x] = GPU_A[row][step * BLOCK_SIZE_XY_K3 + threadIdx.x];
    else
      shared_A_block[threadIdx.y][threadIdx.x] = 0;
    
    if (col < SIZE && (step * BLOCK_SIZE_XY_K3 + threadIdx.y < SIZE))
      shared_B_block[threadIdx.y][threadIdx.x] = GPU_B[step * BLOCK_SIZE_XY_K3 + threadIdx.y][col];
    else
      shared_B_block[threadIdx.y][threadIdx.x] = 0;
    __syncthreads();

    // Partial matrix product
    if (row < SIZE && col < SIZE) {
      for (int k = 0; k < BLOCK_SIZE_XY_K3; k++) {
        shared_C_block[threadIdx.y][threadIdx.x] += shared_A_block[threadIdx.y][k] * shared_B_block[k][threadIdx.x];
      }
    }
    __syncthreads();
  }

  if (row < SIZE && col < SIZE) {
    //Storing results in global memory
    GPU_C[row][col] = shared_C_block[threadIdx.y][threadIdx.x];
  }
}

__global__ void MatrixProductKernel_v3_2(void)
{
  int nbSteps = gridDim.x;
  int row = blockIdx.y*BLOCK_SIZE_XY_K3 + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_XY_K3 + threadIdx.x;

  // Shared memory arrays
  __shared__ T_real shared_A_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  __shared__ T_real shared_B_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  __shared__ T_real shared_C_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  shared_C_block[threadIdx.y][threadIdx.x] = 0;

  // Matrix product computation
  for (int step = 0; step < nbSteps; step++) {
    int start = step * BLOCK_SIZE_XY_K3;

    // RAM to shared memory
    if (row < SIZE && (start + threadIdx.x < SIZE))
      shared_A_block[threadIdx.y][threadIdx.x] = GPU_A[row][start + threadIdx.x];
    // else
    //   shared_A_block[threadIdx.y][threadIdx.x] = 0;
    
    if (col < SIZE && (start + threadIdx.y < SIZE))
      shared_B_block[threadIdx.y][threadIdx.x] = GPU_B[start + threadIdx.y][col];
    // else
    //   shared_B_block[threadIdx.y][threadIdx.x] = 0;
    __syncthreads();

    // Partial matrix product
    if (row < SIZE && col < SIZE) {
      for (int k = 0; k < BLOCK_SIZE_XY_K3; k++) {
        if (k + start < SIZE)
          shared_C_block[threadIdx.y][threadIdx.x] += shared_A_block[threadIdx.y][k] * shared_B_block[k][threadIdx.x];
      }
    }
    __syncthreads();
  }

  if (row < SIZE && col < SIZE) {
    //Storing results in global memory
    GPU_C[row][col] = shared_C_block[threadIdx.y][threadIdx.x];
  }
}

__global__ void MatrixProductKernel_v3_3(void)
{
  int nbSteps = gridDim.x;
  int row = blockIdx.y*BLOCK_SIZE_XY_K3 + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE_XY_K3 + threadIdx.x;

  // Shared memory arrays
  __shared__ T_real shared_A_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  __shared__ T_real shared_B_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  __shared__ T_real shared_C_block[BLOCK_SIZE_XY_K3][BLOCK_SIZE_XY_K3];
  shared_C_block[threadIdx.y][threadIdx.x] = 0;

  // Matrix product computation
  for (int step = 0; step < nbSteps; step++) {
    int start = step * BLOCK_SIZE_XY_K3;

    // RAM to shared memory
    if (row < SIZE && (start + threadIdx.x < SIZE))
      shared_A_block[threadIdx.y][threadIdx.x] = GPU_A[row][start + threadIdx.x];
    // else
    //   shared_A_block[threadIdx.y][threadIdx.x] = 0;
    
    if (col < SIZE && (start + threadIdx.y < SIZE))
      shared_B_block[threadIdx.y][threadIdx.x] = GPU_B[start + threadIdx.y][col];
    // else
    //   shared_B_block[threadIdx.y][threadIdx.x] = 0;
    __syncthreads();

    // Partial matrix product
    if (row < SIZE && col < SIZE) {
      for (int k = 0; k < BLOCK_SIZE_XY_K3; k++) {
        if (k + start >= SIZE)
          break;
        shared_C_block[threadIdx.y][threadIdx.x] += shared_A_block[threadIdx.y][k] * shared_B_block[k][threadIdx.x];
      }
    }
    __syncthreads();
  }

  if (row < SIZE && col < SIZE) {
    //Storing results in global memory
    GPU_C[row][col] = shared_C_block[threadIdx.y][threadIdx.x];
  }
}


/*-------------------------------------------------------------------------------*/
/* Transposition kernel using global memory and registers (slow version)         */
/*-------------------------------------------------------------------------------*/
__global__ void TransposeKernel_v0(T_real *MT, T_real *M, int mRow, int nCol)
{
 int row = threadIdx.y + blockIdx.y*BLOCK_SIZE_XY_KT0;
 int col = threadIdx.x + blockIdx.x*BLOCK_SIZE_XY_KT0;
 
 if (row < mRow && col < nCol)
   MT[col*mRow + row] = M[row*nCol + col];
}


/*-------------------------------------------------------------------------------*/
/* Small matrix product on the local GPU.                                        */
/*-------------------------------------------------------------------------------*/
void gpuProduct(gkid_t kid)
{
 dim3 Dg = {0,0,0};   // Grid descriptor
 dim3 Db = {0,0,0};   // Block descriptor
 
 //T_real alpha;      // When using CUBLAS
 //T_real beta;       // When using CUBLAS

 switch(kid) {

 case GK0 : // Kernel v0 - 1D kernel using only resgisters and cache with generic matrix size
   // - init the grid of blocs
   Db.x = BLOCK_SIZE_X_K0;
   Db.y = 1;
   Db.z = 1;
   Dg.x = (SIZE - 1)/BLOCK_SIZE_X_K0 + 1;
   Dg.y = SIZE;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v0<<<Dg,Db>>>();
   break;

 case GK1 : // kernel v1 : 2D kernel using only registers and cache with generic matrix size
   // - init the grid of blocs
   Db.x = BLOCK_SIZE_X_K1;
   Db.y = BLOCK_SIZE_Y_K1;
   Db.z = 1;
   Dg.x = (SIZE -1)/BLOCK_SIZE_X_K1 +1;
   Dg.y = (SIZE -1)/BLOCK_SIZE_Y_K1 +1;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v1<<<Dg,Db>>>();
   break;

 case GK2 : // kernel v2 : 2D kernel using the shared memories
   // - init the grid of blocs
   Db.x = BLOCK_SIZE_XY_K2;
   Db.y = BLOCK_SIZE_XY_K2;
   Db.z = 1;
   Dg.x = SIZE/BLOCK_SIZE_XY_K2;
   Dg.y = SIZE/BLOCK_SIZE_XY_K2;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v2<<<Dg,Db>>>();
   break;
  
 case GK3 : // kernel v3 : 2D kernel using the shared memories with generic matrix size
    // - init the grid of blocs
   Db.x = BLOCK_SIZE_XY_K3;
   Db.y = BLOCK_SIZE_XY_K3;
   Db.z = 1;
   Dg.x = (SIZE - 1)/BLOCK_SIZE_XY_K3 + 1;
   Dg.y = (SIZE - 1)/BLOCK_SIZE_XY_K3 + 1;
   Dg.z = 1;
   // - run the Grid of Blocs of threads
   MatrixProductKernel_v3<<<Dg,Db>>>();
   break;

 case GK4 : // calling cublas gemm & user-defined transpose kernel
   break;
   
 case GK5 : // Calling cublas gemm & cublas geam kernels
   break;

 case GK6 : // Calling cublas gemm, using matrix math properties
   break;

 case GK7 : // Calling cublas gemmEx with std 32F datatypes, using Tensor cores
   break;

 case GK8 : // Calling cublas gemmEx with low precision on 32 bits datatypes, using Tensor cores
   break;

 default :
   fprintf(stderr,"Unknown GPU kernel!");
   exit(EXIT_FAILURE);
 } // End of switch
}




